#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void AddIntsCUDA(int *a, int *b)
{
for (int i = 0; i < 10000005; i++)
a[0] += b[0];
}

int main()
{
int h_a = 0, h_b = 1;

int *d_a, *d_b;

if (hipMalloc((void**)&d_a, sizeof(int)) != hipSuccess)
{
cout << "Error allocating memory!" << endl;
return 0;
}

if (hipMalloc(&d_b, sizeof(int)) != hipSuccess)
{
cout << "Error allocating memory!" << endl;
hipFree(d_a);
return 0;
}

if (hipMemcpy(d_a, &h_a, sizeof(int), hipMemcpyHostToDevice) != hipSuccess)
{
cout << "Error copying memory!" << endl;
hipFree(d_a);
hipFree(d_b);
return 0;
}

if (hipMemcpy(d_b, &h_b, sizeof(int), hipMemcpyHostToDevice) != hipSuccess)
{
cout << "Error copying memory!" << endl;
hipFree(d_a);
hipFree(d_b);
return 0;
}

AddIntsCUDA<<<1, 1>>>(d_a, d_b);

if (hipMemcpy(&h_a, d_a, sizeof(int), hipMemcpyDeviceToHost) != hipSuccess)
{
cout << "Error copying memory!" << endl;
return 0;
}

cout << h_a << endl;

hipFree(d_a);
hipFree(d_b);

hipDeviceReset();

return 0;
}
