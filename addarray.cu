#include <iostream>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <ctime>

using namespace std;

__global__ void AddIntsCUDA(int *a, int *b, int count)
{
int id = blockIdx.x*blockDim.x + threadIdx.x;

if (id < count)
{
a[id] += b[id];
}
}

int main()
{
srand(time(NULL));

int count = 100;

int *h_a = new int[count];

int *h_b = new int[count];

for (int i = 0; i < count; i++)
{
h_a[i] = rand()%1000;
h_b[i] = rand()%1000;
}

cout << "before add" << endl;

for (int i = 0; i < 5; i++)
cout << h_a[i] << " " << h_b[i] << endl;

int *d_a, *d_b;

if (hipMalloc(&d_a, sizeof(int)*count) != hipSuccess)
{
cout << "Error allocating memory!" << endl;
return 0;
}

if (hipMalloc(&d_b, sizeof(int)*count) != hipSuccess)
{
cout << "Error allocating memory!" << endl;
hipFree(d_a);
return 0;
}

if (hipMemcpy(d_a, h_a, sizeof(int)*count, hipMemcpyHostToDevice) != hipSuccess)
{
cout << "Error copying memory!" << endl;
hipFree(d_a);
hipFree(d_b);
return 0;
}

if (hipMemcpy(d_b, h_b, sizeof(int)*count, hipMemcpyHostToDevice) != hipSuccess)
{
cout << "Error copying memory!" << endl;
hipFree(d_a);
hipFree(d_b);
return 0;
}

AddIntsCUDA<<<count/256 + 1, 256>>>(d_a, d_b, count);

if (hipMemcpy(h_a, d_a, sizeof(int)*count, hipMemcpyDeviceToHost) != hipSuccess)
{
delete[] h_a;
delete[] h_b;
hipFree(d_a);
hipFree(d_b);
cout << "Error copying memory!" << endl;
return 0;
}

for (int i = 0; i < 5; i++)
cout << h_a[i] << endl;

//cout << a << endl;

hipFree(d_a);
hipFree(d_b);
delete[] h_a;
delete[] h_b;

hipDeviceReset();

return 0;
}
