#include <iostream>
//#include <conio.h>
#include <hip/hip_runtime.h>

using namespace std;

struct phew
{
float x, y, z, w;
//float[128];
float padding;
};

__global__ void Mykernel(unsigned long long *time)
{
//__shared__ float shared[1024];
__shared__ phew shared[1024];

unsigned long long startTime = clock();

//shared[0]++;
//shared[threadIdx.x]++;
//shared[threadIdx.x*2]++;
//shared[threadIdx.x*8]++;
//shared[threadIdx.x*32]++;

shared[threadIdx.x].x++;

unsigned long long finishTime = clock();
*time = (finishTime - startTime);
}

int main()
{
unsigned long long time;
unsigned long long *d_time;

hipMalloc(&d_time, sizeof(unsigned long long));

for (int i = 0; i < 10; i++)
{
Mykernel<<<1, 32>>>(d_time);

hipMemcpy(&time, d_time, sizeof(unsigned long long), hipMemcpyDeviceToHost);

cout << (time-14)/32 << endl << endl;
}

hipFree(d_time);

//_getch();
hipDeviceReset();

return 0;
}
